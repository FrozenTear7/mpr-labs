#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <windows.h>
#include "helper_timer.h"

__global__ void add (int *a, int *b, int *c, int n, int threads) {
    int tid = blockIdx.x * threads + threadIdx.x;

    if(tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

void addCpu(int *a, int *b, int *c, int n) {
    int i;

    for (i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

bool vectorCheck(int *a, int *b, int n) {
    int i;

    for (i = 0; i < n; i++) {
        if(a[i] != b[i]) {
            printf("UH OH mismatch %d, %d\n", a[i], b[i]);
            return false;
        }
    }

    return true;
}

int main(void) {
    // Time variables

    // GPU time
    StopWatchInterface *timer;

    // CPU time - for Windows
    LARGE_INTEGER frequency;
    LARGE_INTEGER start;
    LARGE_INTEGER end;
    double interval;

    while(true) {
        int n, blocks, threads;

        printf("Provide n:\n");
        scanf("%d", &n);

        if(n <= 0) {
            printf("n has to be bigger than 0\n");
            continue;
        }

        printf("Provide number of blocks:\n");
        scanf("%d", &blocks);

        if(n <= 1) {
            printf("number of blocks has to be bigger than 1\n");
            continue;
        }

        printf("Provide number of threads:\n");
        scanf("%d", &threads);

        if(n <= 1) {
            printf("number of threads has to be bigger than 1\n");
            continue;
        }

        if(blocks * threads != n) {
            printf("Blocks * threads have to be equal to n\n");
            continue;
        }

        // Other
        int *a = (int*) malloc(n * sizeof(int));
        int *b = (int*) malloc(n * sizeof(int));
        int *c = (int*) malloc(n * sizeof(int));
        int *d = (int*) malloc(n * sizeof(int));

        int *dev_a, *dev_b, *dev_c;

        hipMalloc((void**) &dev_a, n * sizeof(int));
        hipMalloc((void**) &dev_b, n * sizeof(int));
        hipMalloc((void**) &dev_c, n * sizeof(int));
        
        for (int i = 0; i < n; i++) {
            a[i] = i;
            b[i] = i*2;
        }
        
        hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_c, c, n * sizeof(int), hipMemcpyHostToDevice);
        
        // GPU part

        timer=NULL;
        sdkCreateTimer(&timer);
        sdkResetTimer(&timer);
        sdkStartTimer(&timer);

        add <<<blocks, threads>>> (dev_a, dev_b, dev_c, n, threads);

        hipDeviceSynchronize();
        sdkStopTimer(&timer);
        float time = sdkGetTimerValue(&timer);
        sdkDeleteTimer(&timer);

        hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);
        
        // CPU part

        QueryPerformanceFrequency(&frequency);
        QueryPerformanceCounter(&start);

        addCpu(a, b, d, n);

        QueryPerformanceCounter(&end);
        interval = (double) (end.QuadPart - start.QuadPart) / frequency.QuadPart * 1000;

        for (int i = 0; i < n; i++) {
            // printf("%d+%d=%d, %d\n", a[i], b[i], c[i], d[i]);
        }

        bool sameVectors = vectorCheck(c, d, n);

        if(sameVectors) {
            printf("GPU and CPU vectors are properly added\n");
        } else {
            printf("GPU and CPU vectors are not properly added\n");
        }
        
        printf("Time for the kernel: %f ms\n", time);
        printf("CPU time: %f ms\n", interval);
        
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
    }
    
    return 0;
}