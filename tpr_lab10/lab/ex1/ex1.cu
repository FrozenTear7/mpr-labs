#include "hip/hip_runtime.h"
// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <windows.h>
#include "helper_timer.h"

#define GRID_SIZE 32
#define BLOCK_SIZE 32

typedef struct
{	
	int width;
	int height;
	float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.width = A.width; 
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);

	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.width = B.width; 
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);

	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;	
	size = d_C.width * d_C.height * sizeof(float);

	hipMalloc((void**) &d_C.elements, size);
	
	// call kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // define the block size (what is the best value?) 
    dim3 dimGrid(GRID_SIZE, GRID_SIZE); //  choose grid size depending on problem size 
        
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{	
	// each thread computes one element of C and acumulates results to Cvalue

	float Cvalue = 0; 
	int row = blockIdx.y * blockDim.y + threadIdx.y; 
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	if ((row >= A.height) || (col >= B.width))
	{
		return;
	}

	for (int e = 0; e < A.width; e++) 
	{
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];	
	}

	C.elements[row * C.width + col] = Cvalue;
}

//matrix multiplication CPU
void MatMulCPU(Matrix A, Matrix B, Matrix C)
{	
	for (int i = 0; i < A.width; i++) 
	{
		for (int j = 0; j < B.width; j++) 
		{
			C.elements[i * C.width + j] = 0;
			for (int k = 0; k < A.width; k++) 
			{
				C.elements[i * C.width + j] += A.elements[i * A.width + k] * B.elements[k * B.width + j];
			}
		}
	}
}

bool checkMatrices(Matrix C, Matrix D)
{
	for (int i = 0; i < C.width; i++) 
	{
		for (int j = 0; j < D.width; j++) 
		{
			if (C.elements[i * C.width + j] != D.elements[i * D.width + j])
				return false;
		}
	}

	return true;
}

int main(int argc, char * const argv[])
{	
	if (argc != 4)
	{
		printf("Provide matrix files and width \n");
		return -1;
	}

	char *filename1 = argv[1];
	char *filename2 = argv[2];
	int Width = atoi(argv[3]);

	// Time variables

    // GPU time
    StopWatchInterface *timer;

    // CPU time - for Windows
    LARGE_INTEGER frequency;
    LARGE_INTEGER start;
    LARGE_INTEGER end;
    double interval;
	
	if(Width * Width != GRID_SIZE * GRID_SIZE * BLOCK_SIZE * BLOCK_SIZE)
	{
		printf("Sizes of matrices don't match the grid x block sizes\n");
		return -1;
	}

	Matrix A;
	Matrix B;
	Matrix C;
	Matrix D;
	
	A.width = Width;
	B.width = Width;
	C.width = Width;
	D.width = Width;
	
	A.height = Width;
	B.height = Width;
	C.height = Width;
	D.height = Width;
	
	A.elements = new float[Width * Width];
	B.elements = new float[Width * Width];
	C.elements = new float[Width * Width];
	D.elements = new float[Width * Width];
	
	//fill matrices
	std::ifstream A_input;
	std::ifstream B_input;
	A_input.open(filename1);
	B_input.open(filename2);
	
	float a, b;
	A_input >> a;	
	B_input >> b;	
	int i = 0;

	while (!A_input.eof())
	{	
		A.elements[i] = a;
		B.elements[i] = b;
		A_input >> a;	
		B_input >> b;	
		i += 1;
	}

	A_input.close();
	B_input.close();

	// GPU part

	timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	MatMul(A, B, C);

	hipDeviceSynchronize();
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	// CPU part

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&start);

	MatMulCPU(A, B, D);

	QueryPerformanceCounter(&end);
	interval = (double) (end.QuadPart - start.QuadPart) / frequency.QuadPart * 1000;
	
	if(!checkMatrices(C, D))
	{
		printf("CPU and GPU mul didn't return the same results!\n");
	}
	
	std::ofstream C_output;
	std::ofstream D_output;
	C_output.open("C.txt");
	D_output.open("D.txt");

	for (int i = 0; i < Width; i++)
	{	
		for (int j = 0; j < Width; j++)
		{
			C_output << C.elements[i * Width + j] << "\t";
			D_output << D.elements[i * Width + j] << "\t";
		}

		C_output << endl;
		D_output << endl;
	}

	printf("Time for the kernel: %f ms\n", time);
	printf("CPU time: %f ms\n", interval);

	return 0;
}
	 