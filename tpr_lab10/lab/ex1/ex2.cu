#include "hip/hip_runtime.h"
// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <windows.h>
#include "helper_timer.h"

#define GRID_SIZE 18
#define BLOCK_SIZE 32

typedef struct
{	
	int width;
	int height;
	float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.width = A.width; 
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);

	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.width = B.width; 
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);

	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;	
	size = d_C.width * d_C.height * sizeof(float);

	hipMalloc((void**) &d_C.elements, size);
	
	// call kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // define the block size (what is the best value?) 
    dim3 dimGrid(GRID_SIZE, GRID_SIZE); //  choose grid size depending on problem size 
        
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{	
	// each thread computes one element of C and acumulates results to Cvalue

	float Cvalue = 0; 
	int row = blockIdx.y * blockDim.y + threadIdx.y; 
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// printf("%d - %d\n", row, col);

	if ((row >= A.height) || (col >= B.width))
	{
		return;
	}

	for (int e = 0; e < A.width; e++) 
	{
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];	
	}

	C.elements[row * C.width + col] = Cvalue;
}

//matrix multiplication CPU
void MatMulCPU(Matrix A, Matrix B, Matrix C)
{	
	for (int i = 0; i < A.height; i++) 
	{
		for (int j = 0; j < B.width; j++) 
		{
			printf("%d - %d\n", i, j);
			C.elements[i * C.width + j] = 0;
			for (int k = 0; k < A.width; k++) 
			{
				C.elements[i * C.width + j] += A.elements[i * A.width + k] * B.elements[k * B.width + j];
				// printf("%d, %d, %d, %d, %d, %d\n", k, i * C.width + j, i * A.width + k, k * B.width + j, C.width, C.height);
			}
			printf("xd\n");
		}
	}
}

bool checkMatrices(Matrix C, Matrix D)
{
	for (int i = 0; i < C.width; i++) 
	{
		for (int j = 0; j < D.width; j++) 
		{
			if (C.elements[i * C.width + j] != D.elements[i * D.width + j])
				return false;
		}
	}

	return true;
}

int main(int argc, char * const argv[])
{	
	if (argc != 3)
	{
		printf("Provide matrix files \n");
		return -1;
	}

	char *filename1 = argv[1];
	char *filename2 = argv[2];

	// Time variables

    // GPU time
    StopWatchInterface *timer;

    // CPU time - for Windows
    LARGE_INTEGER frequency;
    LARGE_INTEGER start;
    LARGE_INTEGER end;
    double interval;

	Matrix A;
	Matrix B;
	Matrix C;
	Matrix D;
	
	A.width = 640;
	B.width = 480;
	C.width = 640;
	D.width = 640;
	
	A.height = 480;
	B.height = 320;
	C.height = 320;
	D.height = 320;
	
	A.elements = new float[A.width * A.height * 2];
	B.elements = new float[B.width * B.height * 2];
	C.elements = new float[C.width * C.height * 2];
	D.elements = new float[D.width * D.height * 2];
	
	//fill matrices
	std::ifstream A_input;
	std::ifstream B_input;
	A_input.open(filename1);
	B_input.open(filename2);
	
	float a, b;
	A_input >> a;	
	B_input >> b;	
	int i = 0;

	while (!A_input.eof())
	{	
		A.elements[i] = a;
		A_input >> a;	
		i += 1;
	}

	i = 0;

	while (!B_input.eof())
	{	
		B.elements[i] = b;
		B_input >> b;	
		i += 1;
	}

	A_input.close();
	B_input.close();

	// GPU part

	timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	MatMul(A, B, C);

	hipDeviceSynchronize();
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	// CPU part

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&start);

	MatMulCPU(A, B, D);

	QueryPerformanceCounter(&end);
	interval = (double) (end.QuadPart - start.QuadPart) / frequency.QuadPart * 1000;
	
	if(!checkMatrices(C, D))
	{
		printf("CPU and GPU mul didn't return the same results!\n");
	}
	
	std::ofstream C_output;
	std::ofstream D_output;
	C_output.open("C.txt");
	D_output.open("D.txt");

	for (int i = 0; i < C.width; i++)
	{	
		for (int j = 0; j < C.height; j++)
		{
			C_output << C.elements[i * C.width + j] << "\t";
			D_output << D.elements[i * C.width + j] << "\t";
		}

		C_output << endl;
		D_output << endl;
	}

	printf("Time for the kernel: %f ms\n", time);
	printf("CPU time: %f ms\n", interval);

	return 0;
}
	 